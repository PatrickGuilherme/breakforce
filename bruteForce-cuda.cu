#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>

//97 to 122 use only lowercase letters
//65 to 90 use only capital letters
//48 to 57 use only numbers

#define START_CHAR 48
#define END_CHAR 122
#define MAXIMUM_PASSWORD 20

__device__ long long my_pow(long long x, int y)
{
  long long res = 1;
  if (y==0)
    return res;
  else
    return x * my_pow(x, y-1);
}

__device__ int my_strlen(char *s)
{
    int sum = 0;
    while (*s++) sum++;
    return sum;
 }

__global__ void bruteForce(char *pass) 
{
  int pass_b26[MAXIMUM_PASSWORD];
    
  long long int j = blockIdx.x * blockDim.x + threadIdx.x;
  long long int pass_decimal = 0;
  int base = END_CHAR - START_CHAR + 2;
  int found = 0;

  //tamanho da senha
  int size = my_strlen(pass);

  for(int i = 0; i < size; i++)
    pass_b26[i] = (int) pass[i] - START_CHAR + 1; 

  for(int i = size - 1; i > -1; i--)
    pass_decimal += (long long int) pass_b26[i] * my_pow(base, i);

  long long int max = my_pow(base, size);
  char s[MAXIMUM_PASSWORD];

  while(j < max){
    if(found == 1) printf("%lli\n", j);
    
    if(j == pass_decimal){
      printf("Found password!\n");
      int index = 0;

      printf("Password in decimal base: %lli\n", j);
      while(j > 0){
        s[index++] = START_CHAR + j%base-1;
        j /= base;
      }
      s[index] = '\0';
      printf("Found password: %s\n", s);
      found = 1;
      break;
    }
    j += blockDim.x * gridDim.x;
  }
}

int main(int argc, char **argv) 
{
  char password[MAXIMUM_PASSWORD], *password_d;
  
  strcpy(password, argv[1]);
  hipMalloc( (void**)&password_d, MAXIMUM_PASSWORD * sizeof(char));
  hipMemcpy(password_d, password, MAXIMUM_PASSWORD * sizeof(char), hipMemcpyHostToDevice);
  
  double dif, speedup, x;
  time_t t1, t2;

  int deviceId, numberOfSMs;
  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

  int number_of_blocks = numberOfSMs * 32;
  int threads_per_block = atoi(argv[2]);
  printf("number_of_blocks: %d | threads_per_block: %d\n", number_of_blocks, threads_per_block);

  t1 = time(nullptr);
  printf("Try to broke the password: %s\n", password);
  bruteForce<<< number_of_blocks, threads_per_block >>>(password_d);
  hipDeviceSynchronize();
  t2 = time(nullptr);

  dif = difftime (t2, t1);

  printf("\n%f seconds\n", dif);

  FILE *fptr;
  FILE *fptr1;
  char c[1000];

  if ((fptr1 = fopen("firstValue.dat", "r")) != NULL)
  {
    fscanf(fptr1, "%[^\n]", c);
    x = atof(c);
    
    if(dif != 0) speedup = x/dif;
    else speedup = 100;
    
    printf("\n%lf\n", speedup);
    
    fclose(fptr1);
  }

  if ((fptr = fopen("speedup_cuda.dat", "a+")) != NULL)
  {
    fprintf(fptr, "%d\t%1.2f\n", threads_per_block, speedup);
    fclose(fptr);
  }
  else{
    fopen("speedup_cuda.dat", "w+");
    fprintf(fptr, "%d\t%1.2f\n", threads_per_block, speedup);
    fclose(fptr);
  }

  hipFree(password_d);
  return 0;
}